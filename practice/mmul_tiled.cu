#include "hip/hip_runtime.h"
// This program computes a simple version of matrix multiplication
// By: Nick from CoffeeBeforeArch

#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;

#define SHMEM_SIZE = 16 * 16;

__global__ void matrixMul(const int *a, const int *b, int *c, int N, int tile_size) {
  // Statically define A, B tiles
  __shared__ int s_a[SHMEM_SIZE];
  __shared__ int s_b[SHMEM_SIZE];

  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  
  // aliases 
  const int tx = threadIdx.x, bIdxX = blockIdx.x;
  const int ty = threadIdx.y, bIdxY = blockIdx.y;
  const bDim = blockDim.x;

  // iterate over tiles instead of elements
  for (int i=0; i<N; i+=bDim) {
    // get tile: a => sweep right, b => sweep down
    s_a[ty * bDim + tx] = a[(row*N) + (i*bDim + tx)];
    s_b[ty * bDim + tx] = b[(i*bDim*N + ty*N) + (col)];
  }

  __syncthreads();

  // Now go through tile
  int prod = 0;
  for (int i=0; i<N; ++i) {
    prod += a[row*N + i] * b[i*N + col];
  }

  __syncthreads();

  // Iterate over row, and down column
  c[row * N + col] = prod;
  
}

// Check result on the CPU
void verify_result(vector<int> &a, vector<int> &b, vector<int> &c, int N) {
  // For every row...
  for (int i = 0; i < N; i++) {
    // For every column...
    for (int j = 0; j < N; j++) {
      // For every element in the row-column pair
      int tmp = 0;
      for (int k = 0; k < N; k++) {
        // Accumulate the partial results
        tmp += a[i * N + k] * b[k * N + j];
      }

      // Check against the CPU result
      assert(tmp == c[i * N + j]);
    }
  }
}

int main() {
  // Matrix size of 1024 x 1024;
  int N = 1 << 10;

  // Size (in bytes) of matrix
  size_t bytes = N * N * sizeof(int);

  // Host vectors
  vector<int> h_a(N * N);
  vector<int> h_b(N * N);
  vector<int> h_c(N * N);

  // Initialize matrices
  generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
  generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

  // Allocate device memory
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  // Copy data to the device
  hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

  // Threads per CTA dimension
  int THREADS = 32;

  // Blocks per grid dimension (assumes THREADS divides N evenly)
  int BLOCKS = N / THREADS;

  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  // Launch kernel
  matrixMul<<<blocks, threads>>>(d_a, d_b, d_c, N);

  // Copy back to the host
  hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

  // Check result
  verify_result(h_a, h_b, h_c, N);

  cout << "COMPLETED SUCCESSFULLY\n";

  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
