#include "hip/hip_runtime.h"
// This program implements a 1D convolution using CUDA,
// and stores the kernel in constant memory, and loads
// reused values into shared memory (scratchpad)

# include <iostream>
# include <algorithm>
# include <iostream>
# include <vector>

// !!Diff: store in constant memory
extern __constant__ vector<int> s_array = {1,2,1};
#define K_W = s_array.size();

__global__ void conv1dck(int *image, int *kernel, int *output, const int &K_IMG, const int &K_W) {
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    // !!Diff: store in shared memory
    extern __shared__ vector<int> s_array;
    int R = K_W / 2;
    int start_idx = tid - R;
    int prod;
    for (int k=0; k<K_W; ++k) {
        if (start_idx + k >= 0 && start_idx + k < s_array.size()) {
            prod += image[start_idx + k] * kernel[k];
    }
    output[tid] = prod;
    }

}

int main() {
    /*
    get image (and kernel? since it is declared in constant mem)
    compute space the vecs will take size_vec * size_int
    
    create *pointers in gpu
    malloc at &pointers for the vecs

    memcpy vecs from host to device
    fn<<<>>>(args)
    
    memcpy output vec from device to host
    */

    const int IMG_H = image.size(), K_H = kernel.size();
    const int bytes_image = IMG_H * sizeof(int);
    const int bytes_kernel = K_H * sizeof(int);
    const int *d_image, *d_kernel, *d_out;
    
    hipMalloc(&d_image, bytes_image);
    hipMalloc(&d_kernel, bytes_kernel);
    hipMalloc(&d_out, bytes_image);
    std::vector<int> h_result(IMG_H);

    hipMemcpy(d_image, image, bytes_image, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, bytes_kernel, hipMemcpyHostToDevice);


    const int THREADSPERBLOCK = 256, NUMBLOCKS = IMG_H / THREADSPERBLOCK;
    conv1dck<<<THREADSPERBLOCK, NUMBLOCKS>>>(d_image, d_kernel, d_out);

    hipMemcpy(h_result.data(), d_result, bytes_image, hipMemcpyDeviceToHost);

    hipFree(d_image);
    hipFree(d_kernel);
    hipFree(d_out);
    delete[] h_result;

    return 0;
}